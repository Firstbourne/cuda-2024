#include "hip/hip_runtime.h"
﻿// Copyright 2024 Nedelin Dmitry

#define _USE_MATH_DEFINES

#include <math.h>
#include <iostream>

#include "gelu_cuda.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void GeluKernel(const float* input_data, float* output_data, size_t num_elements) {
    size_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_idx >= num_elements) {
        return;
    }
    const float GELU_COEFFICIENT = 0.044715f;
    const float PI_COEFFICIENT = sqrtf(2.0f / M_PI);

    float input_value = input_data[thread_idx];
    output_data[thread_idx] = 0.5f * input_value * (1.0f + tanhf(PI_COEFFICIENT * (input_value + GELU_COEFFICIENT * input_value * input_value * input_value)));
}

std::vector<float> GeluCUDA(const std::vector<float>& host_input) {
    size_t num_elements = host_input.size();

    std::vector<float> host_output(num_elements);
    float* device_input_data = nullptr;
    float* device_output_data = nullptr;
    size_t memory_size = num_elements * sizeof(float);

    hipMalloc(&device_input_data, memory_size);
    hipMalloc(&device_output_data, memory_size);

    hipDeviceProp_t device_properties;
    hipGetDeviceProperties(&device_properties, 0);

    hipMemcpy(device_input_data, host_input.data(), memory_size, hipMemcpyHostToDevice);

    auto threads_per_block = device_properties.maxThreadsPerBlock;
    auto blocks_per_grid = (num_elements + threads_per_block - 1) / threads_per_block;

    GeluKernel << <blocks_per_grid, threads_per_block >> > (device_input_data, device_output_data, num_elements);
    hipMemcpy(host_output.data(), device_output_data, memory_size, hipMemcpyDeviceToHost);

    hipFree(device_input_data);
    hipFree(device_output_data);

    return host_output;
}
