// Copyright (c) 2024 Nedelin Dmitry

#include <iostream>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "gemm_cublas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& matrixA,
                              const std::vector<float>& matrixB,
                              int dimension)
{
    const float scalarAlpha = 1.0f;
    const float scalarBeta = 0.0f;

    const auto totalElements = dimension * dimension;
    std::vector<float> resultMatrix(totalElements);
    const auto totalBytes = totalElements * sizeof(float);

    float* deviceMatrixA = nullptr;
    hipMalloc(&deviceMatrixA, totalBytes);
    float* deviceMatrixB = nullptr;
    hipMalloc(&deviceMatrixB, totalBytes);
    float* deviceMatrixC = nullptr;
    hipMalloc(&deviceMatrixC, totalBytes);

    hipMemcpy(deviceMatrixA, matrixA.data(), totalBytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, matrixB.data(), totalBytes, hipMemcpyHostToDevice);

    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    hipblasSetMathMode(cublasHandle, HIPBLAS_TF32_TENSOR_OP_MATH);

    hipblasGemmEx(cublasHandle,
                 HIPBLAS_OP_N, HIPBLAS_OP_N,
                 dimension, dimension, dimension,
                 &scalarAlpha,
                 deviceMatrixB, HIP_R_32F, dimension,
                 deviceMatrixA, HIP_R_32F, dimension,
                 &scalarBeta,
                 deviceMatrixC, HIP_R_32F, dimension,
                 HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);

    hipMemcpy(resultMatrix.data(), deviceMatrixC, totalBytes, hipMemcpyDeviceToHost);

    hipblasDestroy(cublasHandle);
    hipFree(deviceMatrixC);
    hipFree(deviceMatrixB);
    hipFree(deviceMatrixA);

    return resultMatrix;
}
