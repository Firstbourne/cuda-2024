// Copyright 2024 Kachalov Mikhail
#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float> &a,
                              const std::vector<float> &b,
                              int n)
{
    size_t size = n * n * sizeof(float);
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float *d_b_t;
    hipMalloc(&d_b_t, size);

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, d_b, n, &beta, d_b, n, d_b_t, n);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, n, &alpha, d_a, n, d_b_t, n, &beta, d_c, n);

    std::vector<float> c(n * n);
    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_b_t);
    hipFree(d_c);

    hipblasDestroy(handle);

    return c;
}