// Copyright 2024 Kachalov Mikhail
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <iostream>

std::vector<float> FffCUFFT(const std::vector<float> &input, int batch)
{
    int n = input.size() / (2 * batch);
    std::vector<float> output(input.size(), 0.0f);

    hipfftHandle plan;
    hipfftComplex *d_input, *d_output;
    hipMalloc((void **)&d_input, input.size() * sizeof(float));
    hipMalloc((void **)&d_output, input.size() * sizeof(float));
    hipMemcpy(d_input, input.data(), input.size() * sizeof(float), hipMemcpyHostToDevice);
    hipfftPlanMany(&plan, 1, &n, NULL, 1, n, NULL, 1, n, HIPFFT_C2C, batch);

    hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD);
    hipfftExecC2C(plan, d_output, d_output, HIPFFT_BACKWARD);
    hipMemcpy(output.data(), d_output, output.size() * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < output.size(); i++)
    {
        output[i] /= n;
    }

    hipfftDestroy(plan);
    hipFree(d_input);
    hipFree(d_output);

    return output;
}