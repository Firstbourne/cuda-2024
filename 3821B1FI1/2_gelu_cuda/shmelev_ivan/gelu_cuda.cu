#include "hip/hip_runtime.h"
// Copyright (c) 2024 Shmelev Ivan
#include "gelu_cuda.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cmath>

__global__ void geluKernel(const float* input, float* result, size_t dataSize) {

    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const float geluConstant = 0.044715f;

    if (threadIndex < dataSize) {
        float x = input[threadIndex];
        result[threadIndex] = 0.5f * x * (1.0f + tanh(sqrt(2.0f / M_PI)
         * (x + geluConstant * x * x * x)));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    size_t vectorSize = input.size();
    std::vector<float> result(vectorSize);
    size_t sizeInBytes = vectorSize * sizeof(*input.data());

    float* deviceInput;
    float* deviceOutput;
    hipMalloc(&deviceInput, sizeInBytes);
    hipMalloc(&deviceOutput, sizeInBytes);

    hipMemcpy(deviceInput, input.data(), sizeInBytes, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, 0);
    size_t threadsBlock = deviceProps.maxThreadsPerBlock;
    size_t blocksGrid = (vectorSize + threadsBlock - 1) / threadsBlock;

    geluKernel<<<blocksGrid, threadsBlock>>>(deviceInput, deviceOutput, vectorSize);
    hipMemcpy(result.data(), deviceOutput, sizeInBytes, hipMemcpyDeviceToHost);

    hipFree(deviceInput);
    hipFree(deviceOutput);

    return result;
}