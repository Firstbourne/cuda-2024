#include "hip/hip_runtime.h"
/*When entering the following vector:
int n = 4;
    std::vector<float> a = {1.0, 2.0, 3.0, 4.0,
                            5.0, 6.0, 7.0, 8.0,
                            9.0, 10.0, 11.0, 12.0,
                            13.0, 14.0, 15.0, 16.0};
    std::vector<float> b = {1.0, 0.0, 0.0, 0.0,
                            0.0, 1.0, 0.0, 0.0,
                            0.0, 0.0, 1.0, 0.0,
                            0.0, 0.0, 0.0, 1.0};
The output values ​​were:
1 2 3 4 
5 6 7 8 
9 10 11 12 
13 14 15 16 
*/

#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

__global__ void gemmKernel(const float* a, const float* b, float* c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    std::vector<float> c(n * n, 0.0f);

    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_c = nullptr;

    hipMalloc(&d_a, n * n * sizeof(float));
    hipMalloc(&d_b, n * n * sizeof(float));
    hipMalloc(&d_c, n * n * sizeof(float));

    hipMemcpy(d_a, a.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), n * n * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

    gemmKernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy(c.data(), d_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}