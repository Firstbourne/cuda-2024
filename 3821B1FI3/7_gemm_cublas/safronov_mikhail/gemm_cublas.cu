#include "gemm_cublas.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
  std::vector<float> c(n * n);
  float* ptr_a;
  float* ptr_b;
  float* ptr_c;
  hipMalloc(&ptr_a, sizeof(float) * n * n);
  hipMalloc(&ptr_b, sizeof(float) * n * n);
  hipMalloc(&ptr_c, sizeof(float) * n * n);
  hipMemcpy(ptr_a, a.data(), sizeof(float) * n * n, hipMemcpyHostToDevice);
  hipMemcpy(ptr_b, b.data(), sizeof(float) * n * n, hipMemcpyHostToDevice);

  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, ptr_b, n, ptr_a, n, &beta, ptr_c, n);
  hipblasDestroy(handle);

  hipMemcpy(c.data(), ptr_c, sizeof(float) * n * n, hipMemcpyDeviceToHost);

  hipFree(ptr_a);
  hipFree(ptr_b);
  hipFree(ptr_c);

  return c;
}
