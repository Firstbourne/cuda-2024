#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(callable)                                  \
  {                                                                 \
    auto codeError = callable;                                      \
    if (codeError != hipSuccess) {                                 \
      std::cerr << "CUDA error: " << hipGetErrorString(codeError)   \
                << '\n';                                             \
      exit(codeError);                                              \
    }                                                               \
  }

__global__ void naive_gemm_kernel(float *c, const float *a, const float *b,
                                  const size_t size) {
  size_t mIdx = blockIdx.y * blockDim.y + threadIdx.y;
  size_t nIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (mIdx < size && nIdx < size) {
    float cVal = 0.0f;
    for (size_t k = 0; k < size; ++k)
        cVal += a[mIdx * size + k] * b[size * k + nIdx];
    c[mIdx * size + nIdx] = cVal;
  }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
    const std::vector<float>& b, int size) {
    std::vector<float> c(size * size);
    
    size_t sizeInBytes = size * size * sizeof(*a.data());
    
    float* d_a;
    hipMalloc(&d_a, sizeInBytes);
    float* d_b;
    hipMalloc(&d_b, sizeInBytes);
    float* d_c;
    hipMalloc(&d_c, sizeInBytes);
    
    hipMemcpy(d_a, a.data(), sizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), sizeInBytes, hipMemcpyHostToDevice);
    
    const size_t sizeAxis = 32u;
    dim3 threadsPerBlock(
        sizeAxis,
        sizeAxis
    );
    dim3 numBlocks(
        (size + sizeAxis - 1) / sizeAxis,
        (size + sizeAxis - 1) / sizeAxis
    );
    
    naive_gemm_kernel<<<numBlocks, threadsPerBlock>>>(d_c, d_a, d_b, size);

    hipMemcpy(c.data(), d_c, sizeInBytes, hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return c;
}