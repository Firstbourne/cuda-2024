#include "hip/hip_runtime.h"
#include "fft_cufft.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>

__global__ void norm_kernel(float* a, const int n, const int nx) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    a[i] /= nx;
  }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
  int device;
  hipGetDevice(&device);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  int threadsPerBlock = deviceProp.maxThreadsPerBlock;
  int blockNum = (input.size() + threadsPerBlock - 1) / threadsPerBlock;

  size_t nx = input.size() / batch / 2;

  hipfftComplex* ptr;
  hipMalloc(&ptr, sizeof(float) * input.size());
  hipMemcpy(ptr, input.data(), sizeof(float) * input.size(), hipMemcpyHostToDevice);

  hipfftHandle plan;
  hipfftPlan1d(&plan, nx, HIPFFT_C2C, batch);
  hipfftExecC2C(plan, ptr, ptr, HIPFFT_FORWARD);
  hipfftExecC2C(plan, ptr, ptr, HIPFFT_BACKWARD);

  std::vector<float> result(input.size());
  norm_kernel<<<blockNum, threadsPerBlock>>>(reinterpret_cast<float*>(ptr), input.size(), nx);
  hipMemcpy(result.data(), ptr, sizeof(float) * input.size(), hipMemcpyDeviceToHost);

  hipfftDestroy(plan);
  hipFree(ptr);

  return result;
}
