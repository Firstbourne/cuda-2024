#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <>
#include <iostream>
#include <vector>
#include <chrono>
#include <iostream>
#include <random>
#include <vector>
#include <cmath>
#include <iomanip>


#define CHECK_CUDA(call)                                                \
    {                                                                   \
        auto err = call;                                                \
        if (err != hipSuccess) {                                       \
            std::cerr << "CUDA Error: " << hipGetErrorString(err)      \
                      << " at " << __FILE__ << ":" << __LINE__ << "\n"; \
            std::exit(err);                                             \
        }                                                               \
    }


#define CHECK_CUFFT(call)                                               \
    {                                                                   \
        auto err = call;                                                \
        if (err != HIPFFT_SUCCESS) {                                     \
            std::cerr << "cuFFT Error: " << static_cast<int>(err)       \
                      << " at " << __FILE__ << ":" << __LINE__ << "\n"; \
            std::exit(err);                                             \
        }                                                               \
    }


__global__ void normalizeKernel(float* data, int size, float normFactor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] *= normFactor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    if (input.empty()) return {};

    const size_t totalSize = input.size();
    const int n = (totalSize / batch) >> 1;  // ���������� ����������� ��������� � ����� ������
    const size_t byteSize = sizeof(hipfftComplex) * n * batch;


    hipDeviceProp_t deviceProp;
    CHECK_CUDA(hipGetDeviceProperties(&deviceProp, 0));
    const int threadsPerBlock = deviceProp.maxThreadsPerBlock;
    const int numBlocks = (totalSize + threadsPerBlock - 1) / threadsPerBlock;

    std::vector<float> output(totalSize);

 
    hipfftComplex* d_data;
    CHECK_CUDA(hipMalloc(&d_data, byteSize));
    CHECK_CUDA(hipMemcpy(d_data, input.data(), byteSize, hipMemcpyHostToDevice));


    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlan1d(&plan, n, HIPFFT_C2C, batch));


    CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));


    CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD));


    const float normFactor = 1.0f / n;
    normalizeKernel<<<numBlocks, threadsPerBlock>>>(
        reinterpret_cast<float*>(d_data), totalSize, normFactor);
    CHECK_CUDA(hipDeviceSynchronize());


    CHECK_CUDA(hipMemcpy(output.data(), d_data, byteSize, hipMemcpyDeviceToHost));


    CHECK_CUFFT(hipfftDestroy(plan));
    CHECK_CUDA(hipFree(d_data));

    return output;
}
