#include "hip/hip_runtime.h"
#include "gelu_cuda.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void GuluCUDA_kernel(float* a, const int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		constexpr float twooverpi = 0.7978845608028653;
		float x = a[i];
		a[i] = 0.5f * x * (1.f + tanhf(twooverpi * x * (1.0f + 0.044715f * x * x)));
	}
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
	int device;
	hipGetDevice(&device);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);
	int threadsPerBlock = deviceProp.maxThreadsPerBlock;
	int blockNum = (input.size() + threadsPerBlock - 1) / threadsPerBlock;
	
	std::vector<float> output(input);
	float* ptr;
	hipMalloc(&ptr, sizeof(float) * input.size());
	hipMemcpy(ptr, output.data(), sizeof(float) * input.size(), hipMemcpyHostToDevice);
	
	GuluCUDA_kernel<<<blockNum, threadsPerBlock>>>(ptr, input.size());
	
	hipMemcpy(output.data(), ptr, sizeof(float) * input.size(), hipMemcpyDeviceToHost);
	hipFree(ptr);
	
	return output;
}
