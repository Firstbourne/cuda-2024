#include "hip/hip_runtime.h"
// Copyright (c) 2024 Zakharov Artem
#include "fft_cufft.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include ""


__global__ void normalize_signal_kernel(float* signal, int size, float k) {
    int ind = blockDim.x * blockIdx.x + threadIdx.x;
    if (ind < size) {
        signal[ind] *= k;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);

    int size = input.size();
    int elements_per_batch = size / batch / 2;
    int bytes_size = sizeof(hipfftComplex) * elements_per_batch * batch;

    int threads_per_block = device_prop.maxThreadsPerBlock;
    int num_blocks = (size + threads_per_block - 1) / threads_per_block;

    std::vector<float> output(size);

    hipfftComplex* signal;
    hipMalloc(reinterpret_cast<void**>(&signal), bytes_size);
    hipMemcpy(reinterpret_cast<void*>(signal),
               reinterpret_cast<const void*>(input.data()),
               bytes_size, hipMemcpyHostToDevice);

    hipfftHandle handle;
    hipfftPlan1d(&handle, elements_per_batch, HIPFFT_C2C, batch);
    hipfftExecC2C(handle, signal, signal, HIPFFT_FORWARD);
    hipfftExecC2C(handle, signal, signal, HIPFFT_BACKWARD);
    hipfftDestroy(handle);

    normalize_signal_kernel<<<num_blocks, threads_per_block>>>(
            reinterpret_cast<float*>(signal), size, 1.0f / elements_per_batch);
    hipMemcpy(reinterpret_cast<void*>(output.data()),
               reinterpret_cast<const void*>(signal),
               bytes_size, hipMemcpyDeviceToHost);
    hipFree(reinterpret_cast<void*>(signal));

    return output;
}
