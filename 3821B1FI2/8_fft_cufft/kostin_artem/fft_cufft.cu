
#include <hip/hip_runtime.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <iostream>

__global__ void normalize_kernel(float* data, size_t size, float norm_factor) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] *= norm_factor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    size_t size = input.size();
    int elemPerBatch = size / (2 * batch);

    hipfftComplex* d_signal = nullptr;
    size_t complex_size = sizeof(hipfftComplex) * elemPerBatch * batch;
    hipMalloc(&d_signal, complex_size);

    hipMemcpy(d_signal, input.data(), complex_size, hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, elemPerBatch, HIPFFT_C2C, batch);
    hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);
    hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_BACKWARD);

    int threadsPerBlock = 256;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    normalize_kernel<<<numBlocks, threadsPerBlock>>>(reinterpret_cast<float*>(d_signal), size, 1.0f / elemPerBatch);

    hipDeviceSynchronize();

    std::vector<float> output(size);
    hipMemcpy(output.data(), d_signal, complex_size, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_signal);

    return output;
}
