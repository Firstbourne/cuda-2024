#include "hip/hip_runtime.h"
// Copyright (c) 2024 Soloninko Andrey
#include <hip/hip_runtime.h>
#include <>

#include <cmath>
#include <iostream>

#include "gelu_cuda.h"

__global__ void GeluKernel(const float* input, float* output, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    float x = input[idx];
    output[idx] = x * 0.5f * (1.0f + tanhf(0.7978845608028654f * (x + 0.044715f * x * x * x)));
  }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
  if (input.empty()) return {};

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);

  size_t size = input.size();
  size_t countBytes = size * sizeof(float);
  
  std::vector<float> output(size);
  
  auto threadsPerBlock = deviceProp.maxThreadsPerBlock;
  auto numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

  float* inpDev = nullptr;
  float* outDev = nullptr;

  hipMalloc(&inpDev, countBytes);
  hipMalloc(&outDev, countBytes);

  hipMemcpy(inpDev, input.data(), countBytes, hipMemcpyHostToDevice);

  GeluKernel<<<numBlocks, threadsPerBlock>>>(inpDev, outDev, size);

  hipMemcpy(output.data(), outDev, countBytes, hipMemcpyDeviceToHost);

  hipFree(inpDev);
  hipFree(outDev);

  return output;
}
