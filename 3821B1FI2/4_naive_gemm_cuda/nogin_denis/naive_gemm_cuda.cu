#include "hip/hip_runtime.h"
// Copyright (c) 2024 Nogin Denis
#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "naive_gemm_cuda.h"

__global__ void MatrixMulKernel(const float* a, const float* b, float* c,
                                int size) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < size && row < size) {
    float sum = 0.0f;
    for (int k = 0; k < size; ++k) {
      sum += a[row * size + k] * b[k * size + col];
    }
    c[row * size + col] = sum;
  }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b, int size) {
  std::vector<float> c(size * size);

  float* d_a = nullptr;
  float* d_b = nullptr;
  float* d_c = nullptr;

  hipMalloc(&d_a, static_cast<unsigned long long>(size) * size * sizeof(float));
  hipMalloc(&d_b, static_cast<unsigned long long>(size) * size * sizeof(float));
  hipMalloc(&d_c, static_cast<unsigned long long>(size) * size * sizeof(float));

  hipMemcpy(d_a, a.data(),
             static_cast<unsigned long long>(size) * size * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(),
             static_cast<unsigned long long>(size) * size * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (size + threadsPerBlock.y - 1) / threadsPerBlock.y);

  MatrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, size);

  hipMemcpy(c.data(), d_c,
             static_cast<unsigned long long>(size) * size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return c;
}
