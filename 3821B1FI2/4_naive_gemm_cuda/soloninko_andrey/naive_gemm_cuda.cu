#include "hip/hip_runtime.h"
// Copyright (c) 2024 Soloninko Andrey
#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "naive_gemm_cuda.h"

__global__ void MatrixMulKernel(const float* a, const float* b, float* c,
                                int n) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n && col < n) {
    float sum = 0.0f;
    for (int k = 0; k < n; ++k) {
      sum += a[row * n + k] * b[k * n + col];
    }
    c[row * n + col] = sum;
  }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b, int n) {
  
  const size_t count = n * n;
  std::vector<float> c(n * n);

  float* d_a;
  float* d_b;
  float* d_c;

  hipMalloc(&d_a, count * sizeof(float));
  hipMalloc(&d_b, count * sizeof(float));
  hipMalloc(&d_c, count * sizeof(float));

  hipMemcpy(d_a, a.data(),
             count * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(),
             count * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

  MatrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, n);

  hipMemcpy(c.data(), d_c,
             count * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return c;
}
