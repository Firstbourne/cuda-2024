#include "gemm_cublas.h"
#include <vector>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {

    std::vector<float> c(n * n);

    float *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, n * n * sizeof(float));
    hipMalloc((void**)&d_b, n * n * sizeof(float));
    hipMalloc((void**)&d_c, n * n * sizeof(float));

    hipMemcpy(d_a, a.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), n * n * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle, 
                HIPBLAS_OP_N, HIPBLAS_OP_N, 
                n, n, n, 
                &alpha, 
                d_b, n, 
                d_a, n, 
                &beta, 
                d_c, n);

    hipMemcpy(c.data(), d_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);

    return c;
}
