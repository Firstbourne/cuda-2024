// Copyright (c) 2024 Zakharov Artem
#include "gemm_cublas.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    int size = n * n;
    std::vector<float> c(size);
    size_t bytes_size = size * sizeof(float);
    float alpha = 1.0;
    float beta = 0.0;

    float *a_dev, *b_dev, *c_dev;

    hipMalloc(reinterpret_cast<void**>(&a_dev), bytes_size);
    hipMalloc(reinterpret_cast<void**>(&b_dev), bytes_size);
    hipMalloc(reinterpret_cast<void**>(&c_dev), bytes_size);

    hipMemcpy(reinterpret_cast<void*>(a_dev),
               reinterpret_cast<const void*>(a.data()),
               bytes_size, hipMemcpyHostToDevice);
    hipMemcpy(reinterpret_cast<void*>(b_dev),
               reinterpret_cast<const void*>(b.data()),
               bytes_size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha,
                b_dev, n, a_dev, n, &beta, c_dev, n);
    hipblasDestroy(handle);

    hipMemcpy(reinterpret_cast<void*>(c.data()),
               reinterpret_cast<const void*>(c_dev),
               bytes_size, hipMemcpyDeviceToHost);

    hipFree(reinterpret_cast<void*>(a_dev));
    hipFree(reinterpret_cast<void*>(b_dev));
    hipFree(reinterpret_cast<void*>(c_dev));

    return c;
}
