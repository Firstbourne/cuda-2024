#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

// CUDA Kernel for matrix multiplication
__global__ void MatrixMultiplyKernel(const float* a, const float* b, float* c, int n) {
    // Calculate the row and column of the current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a, const std::vector<float>& b, int n) {
    // Total size of matrices
    size_t size = n * n * sizeof(float);

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy matrices to device
    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    // Define block size and grid size
    dim3 blockDim(16, 16);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    MatrixMultiplyKernel<<<gridDim, blockDim>>>(d_a, d_b, d_c, n);

    // Wait for the GPU to finish
    hipDeviceSynchronize();

    // Allocate host memory for the result
    std::vector<float> c(n * n);

    // Copy result back to host
    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}
