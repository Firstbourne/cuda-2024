#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

__global__ void MatrixMultiplyKernel(const float* a, const float* b, float* c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a, const std::vector<float>& b, int n) {
    size_t size = n * n * sizeof(float);

    float* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);

    MatrixMultiplyKernel << <gridDim, blockDim >> > (d_a, d_b, d_c, n);

    hipDeviceSynchronize();

    std::vector<float> c(n * n);

    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}