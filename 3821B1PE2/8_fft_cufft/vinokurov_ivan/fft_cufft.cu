#include "hip/hip_runtime.h"
//  Copyright (c) 2024 Vinokurov Ivan
#include <iostream>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fft_cufft.h"

__global__ void normalizeKernel(float* __restrict__ dataPtr, int totalSize, float scaleFactor) {
    const int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    const int vectorIdx = threadId / 4;

    if (vectorIdx < totalSize / 4) {
        float4* vectorData = reinterpret_cast<float4*>(dataPtr);
        float4 vectorElement = __ldg(&vectorData[vectorIdx]);
        vectorElement.x *= scaleFactor;
        vectorElement.y *= scaleFactor;
        vectorElement.z *= scaleFactor;
        vectorElement.w *= scaleFactor;
        vectorData[vectorIdx] = vectorElement;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch)
{
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);

    const int inputSize = input.size();
    std::vector<float> result(inputSize);

    const int elementsPerBatch = inputSize / batch >> 1;
    const int memorySize = sizeof(hipfftComplex) * elementsPerBatch * batch;
    const int threadsPerBlock = deviceProperties.maxThreadsPerBlock;
    const int totalBlocks = (inputSize + threadsPerBlock - 1) / threadsPerBlock;

    hipfftComplex* deviceBuffer;
    hipMalloc(&deviceBuffer, memorySize);
    hipMemcpy(deviceBuffer, input.data(), memorySize, hipMemcpyHostToDevice);

    hipfftHandle fftPlan;
    hipfftPlan1d(&fftPlan, elementsPerBatch, HIPFFT_C2C, batch);
    hipfftExecC2C(fftPlan, deviceBuffer, deviceBuffer, HIPFFT_FORWARD);
    hipfftExecC2C(fftPlan, deviceBuffer, deviceBuffer, HIPFFT_BACKWARD);

    normalizeKernel<<<totalBlocks, threadsPerBlock>>>(reinterpret_cast<float*>(deviceBuffer), inputSize, 1.0f / elementsPerBatch);

    hipMemcpy(result.data(), deviceBuffer, memorySize, hipMemcpyDeviceToHost);
    hipfftDestroy(fftPlan);
    hipFree(deviceBuffer);

    return result;
}
