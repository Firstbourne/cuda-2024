#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

__global__ void normalize(hipfftComplex* data, int num_elements, float norm_factor) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num_elements) {
        data[idx].x *= norm_factor;
        data[idx].y *= norm_factor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() / (2 * batch);

    if (input.size() != 2 * n * batch) {
        throw std::invalid_argument("Input size must be 2 * n * batch, where n is the number of elements per signal.");
    }

    hipfftComplex* d_data;
    size_t input_size = input.size() * sizeof(float);

    hipMalloc(&d_data, input_size);
    hipMemcpy(d_data, input.data(), input_size, hipMemcpyHostToDevice);

    hipfftHandle plan;
    if (hipfftPlan1d(&plan, n, HIPFFT_C2C, batch) != HIPFFT_SUCCESS) {
        hipFree(d_data);
        throw std::runtime_error("CUFFT plan creation failed.");
    }

    if (hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        hipfftDestroy(plan);
        hipFree(d_data);
        throw std::runtime_error("CUFFT execution for forward FFT failed.");
    }

    if (hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        hipfftDestroy(plan);
        hipFree(d_data);
        throw std::runtime_error("CUFFT execution for inverse FFT failed.");
    }

    int num_elements = n * batch;
    float norm_factor = 1.0f / static_cast<float>(n);
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;

    normalize<<<blocksPerGrid, threadsPerBlock>>>(d_data, num_elements, norm_factor);

    std::vector<float> result(input.size());
    hipMemcpy(result.data(), d_data, input_size, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_data);

    return result;
}
