//  Copyright (c) 2024 Vinokurov Ivan
#include <iostream>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "gemm_cublas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n)
{
    const float scalarAlpha = 1.0f;
    const float scalarBeta = 0.0f;

    const auto totalElements = n * n;
    std::vector<float> output(totalElements);
    const auto sizeInBytes = totalElements * sizeof(float);

    float* deviceMtxA = nullptr;
    hipMalloc(&deviceMtxA, sizeInBytes);
    float* deviceMtxB = nullptr;
    hipMalloc(&deviceMtxB, sizeInBytes);
    float* deviceMtxC = nullptr;
    hipMalloc(&deviceMtxC, sizeInBytes);

    hipMemcpy(deviceMtxA, a.data(), sizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceMtxB, b.data(), sizeInBytes, hipMemcpyHostToDevice);

    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    hipblasSetMathMode(cublasHandle, HIPBLAS_TF32_TENSOR_OP_MATH);

    hipblasGemmEx(cublasHandle,
                 HIPBLAS_OP_N, HIPBLAS_OP_N,
                 n, n, n,
                 &scalarAlpha,
                 deviceMtxB, HIP_R_32F, n,
                 deviceMtxA, HIP_R_32F, n,
                 &scalarBeta,
                 deviceMtxC, HIP_R_32F, n,
                 HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);

    hipMemcpy(output.data(), deviceMtxC, sizeInBytes, hipMemcpyDeviceToHost);

    hipblasDestroy(cublasHandle);
    hipFree(deviceMtxC);
    hipFree(deviceMtxB);
    hipFree(deviceMtxA);

    return output;
}
