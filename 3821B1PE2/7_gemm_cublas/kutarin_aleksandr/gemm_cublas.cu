//Copyright Kutarin Aleksandr 2024

#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

// Function to multiply two matrices using cuBLAS
std::vector<float> GemmCUBLAS(const std::vector<float>& a, const std::vector<float>& b, int n) {
    if (a.size() != n * n || b.size() != n * n) {
        throw std::invalid_argument("Matrix dimensions do not match the expected size.");
    }

    // Result matrix
    std::vector<float> c(n * n, 0);

    // Pointers for device memory
    float *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc((void**)&d_a, n * n * sizeof(float));
    hipMalloc((void**)&d_b, n * n * sizeof(float));
    hipMalloc((void**)&d_c, n * n * sizeof(float));

    // Copy data to device
    hipMemcpy(d_a, a.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), n * n * sizeof(float), hipMemcpyHostToDevice);

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // cuBLAS constants for matrix multiplication
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    // Note: cuBLAS expects column-major storage
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_b, n,  // Matrix B
                d_a, n,  // Matrix A
                &beta,
                d_c, n); // Matrix C

    // Copy result back to host
    hipMemcpy(c.data(), d_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);

    return c;
}
