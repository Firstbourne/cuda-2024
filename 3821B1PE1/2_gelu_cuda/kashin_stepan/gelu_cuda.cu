#include "hip/hip_runtime.h"
#include "gelu_cuda.h"

__global__ void GeluKernel(float *input, float *res, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  float x = input[i];
  auto expon = __expf(x * fma(__powf(x, 2.0f), GELU_COEF2, GELU_COEF1));
  
  res[i] = x * (expon / (1.0f + expon));
}

std::vector<float> GeluCUDA(const std::vector<float> &input) {
  auto size = input.size();
  std::vector<float> output(size);
  float *d_input, *d_output;

  hipMalloc(&d_input, input.size() * sizeof(float));
  hipMalloc(&d_output, output.size() * sizeof(float));
  hipMemcpy(d_input, input.data(), size * sizeof(float),
             hipMemcpyHostToDevice);

  int blockSize = 128;
  int numBlocks = (input.size() + blockSize - 1) / blockSize;

  GeluKernel<<<numBlocks, blockSize>>>(d_input, d_output, size);

  hipMemcpy(output.data(), d_output, size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_output);
  hipFree(d_input);
  return output;
}
