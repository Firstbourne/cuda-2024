#include "hip/hip_runtime.h"
// Copyright (c) 2024 Podyachikh Mikhail
#include "gelu_cuda.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void geluCUDA_kernel(float* a, const int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        const float scale = 0.7978845608028653f; // sqrt(2/pi)
        float val = a[idx];
        a[idx] = 0.5f * val * (1.0f + tanhf(scale * val * (1.0f + 0.044715f * val * val)));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    int threadsPerBlock = deviceProp.maxThreadsPerBlock;
    int numBlocks = (input.size() + threadsPerBlock - 1) / threadsPerBlock;

    std::vector<float> output(input);
    float* deviceBuffer = nullptr;

    hipMalloc(&deviceBuffer, input.size() * sizeof(float));
    hipMemcpy(deviceBuffer, output.data(), input.size() * sizeof(float), hipMemcpyHostToDevice);

    geluCUDA_kernel<<<numBlocks, threadsPerBlock>>>(deviceBuffer, input.size());

    hipMemcpy(output.data(), deviceBuffer, input.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(deviceBuffer);

    return output;
}
