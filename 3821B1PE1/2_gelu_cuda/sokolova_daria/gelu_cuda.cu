#include "hip/hip_runtime.h"
// Copyright (c) 2024 Sokolova Daria
#include "gelu_cuda.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>

global void GeluKernel(const float* input, float* output, size_t size) {
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= size) {
    return;
  }

  const float factor = std::sqrt(2.0f / M_PI);
  constexpr float cubicCoeff = 0.044715f;

  float curr = input[index];
  output[index] = 0.5f * curr * (1.0f + std::tanh(factor * (curr + cubicCoeff * curr * curr * curr)));
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
  size_t size = input.size();

  if (size == 0) {
    return {};
  }

  std::vector<float> output(size);
  float* deviceInputArray = nullptr;
  float* deviceOutputArray = nullptr;
  size_t bufferSize = size * sizeof(float);

  hipMalloc(&deviceInputArray, bufferSize);
  hipMalloc(&deviceOutputArray, bufferSize);

  hipMemcpy(deviceInputArray, input.data(), bufferSize, hipMemcpyHostToDevice);

  hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, 0);
  size_t threadsPerBlock = deviceProperties.maxThreadsPerBlock;
  size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  GeluKernel<<<blocksPerGrid, threadsPerBlock>>>(deviceInputArray, deviceOutputArray, size);
  hipMemcpy(output.data(), deviceOutputArray, bufferSize, hipMemcpyDeviceToHost);

  hipFree(deviceInputArray);
  hipFree(deviceOutputArray);

  return output;
}
