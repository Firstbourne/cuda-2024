#include "hip/hip_runtime.h"
// Copyright (c) 2024 Kirillov Maxim
#include <math.h>
#include <iostream>

#include "gelu_cuda.h"
#include "hip/hip_runtime.h"
#include ""


__global__ void GeluKernel(const float* input, float* output, size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= size) {
        return;
    }
    const float geluParameter = 0.044715f;
    const float piParameter = sqrtf(2.0f / M_PI);

    float x = input[index];
    output[index] = 0.5f * x * (1.0f + tanhf(piParameter * (x + geluParameter * x * x * x)));
}


std::vector<float> GeluCUDA(const std::vector<float>& input) {
    if (input.empty()) {
        return {};
    }
    size_t size = input.size();

    std::vector<float> output(size);
    float* deviceInput = nullptr;
    float* deviceOutput = nullptr;
    size_t sizeInBytes = size * sizeof(float);

    hipMalloc(&deviceInput, sizeInBytes);
    hipMalloc(&deviceOutput, sizeInBytes);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    hipMemcpy(deviceInput, input.data(), sizeInBytes, hipMemcpyHostToDevice);

    auto threads_per_block = deviceProp.maxThreadsPerBlock;
    auto blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    GeluKernel<<<blocks_per_grid, threads_per_block >>>(deviceInput, deviceOutput, size);
    hipMemcpy(output.data(), deviceOutput, sizeInBytes, hipMemcpyDeviceToHost);

    hipFree(deviceInput);
    hipFree(deviceOutput);

    return output;
}
