#include "hip/hip_runtime.h"
// Copyright (c) 2024 Tushentsova Karina
#include "gelu_cuda.h"

#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>


__global__ void GeluKernel(const float* input, float* output, size_t size) {
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size) {
      const float sqrtOver = std::sqrt(2.0f / M_PI);
      constexpr float gelu_coeff = 0.044715f;

      float value = input[index];
      float cubValue = value * value * value;
      float tanIn = sqrtOver * (value + gelu_coeff * cubValue);
      output[index] = 0.5f * value * (1.0f + std::tanh(tanIn));
  }
}

std::vector<float> GeluCUDA(const std::vector<float>& input){
  size_t size = input.size();

  if (size == 0) {
    return {};
  }

  std::vector<float> output(size);
  float* deviceIn = nullptr;
  float* deviceOut = nullptr;
  size_t memorySize = size * sizeof(float);

  hipMalloc(&deviceIn, memorySize);
  hipMalloc(&deviceOut, memorySize);

  hipMemcpy(deviceIn, input.data(), memorySize, hipMemcpyHostToDevice);

  hipDeviceProp_t hipDeviceProp_t;
  hipGetDeviceProperties(&hipDeviceProp_t, 0);
  size_t threadsPerBlock = hipDeviceProp_t.maxThreadsPerBlock;
  size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  GeluKernel<<<blocksPerGrid, threadsPerBlock>>>(deviceIn, deviceOut, size);
  hipMemcpy(output.data(), deviceOut, memorySize, hipMemcpyDeviceToHost);

  hipFree(deviceIn);
  hipFree(deviceOut);

  return output;
}