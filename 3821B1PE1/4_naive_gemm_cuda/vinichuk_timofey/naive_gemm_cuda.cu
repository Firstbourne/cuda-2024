#include "hip/hip_runtime.h"
// Copyright (c) 2024 Vinichuk Timofey

#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>

__global__ void NaiveGemmKernel(const float* a, const float* b, float* c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < n && col < n) {
        
        float sum = 0.0;
        for (int k = 0; k < n; ++k) {
             sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
    const std::vector<float>& b,
    int n) {
    std::vector<float> c(n * n, 0.0f);

    float* kernel_a = nullptr;
    float* kernel_b = nullptr;
    float* kernel_c = nullptr;

    hipMalloc(&kernel_a, n * n * sizeof(float));
    hipMalloc(&kernel_b, n * n * sizeof(float));
    hipMalloc(&kernel_c, n * n * sizeof(float));
    hipMemcpy(kernel_a, a.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(kernel_b, b.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(kernel_c, c.data(), n * n * sizeof(float), hipMemcpyHostToDevice);

    size_t size = 32;
    dim3 blockSize(size, size);
    dim3 numBlocks((n + size - 1) / size, (n + size - 1) / size);

    NaiveGemmKernel <<<numBlocks, blockSize >>> (kernel_a, kernel_b, kernel_c, n);
    hipMemcpy(c.data(), kernel_c, n * n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(kernel_a);
    hipFree(kernel_b);
    hipFree(kernel_c);

    return c;
}
