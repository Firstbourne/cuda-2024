#include "hip/hip_runtime.h"
// Copyright (c) 2024 Vinichuk Timofey
#include "fft_cufft.h"

__global__ void normalizeKernel(float* input, int size, int n) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		input[index] /= n;
	}
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
	std::vector<float> output(input.size());

	int n = input.size() / (batch * 2);

	hipfftHandle handle;
	hipfftPlan1d(&handle, n, HIPFFT_C2C, batch);

	hipfftComplex* data;
	hipMalloc(&data, sizeof(hipfftComplex) * n * batch);
	hipMemcpy(
		data,
		input.data(),
		sizeof(hipfftComplex) * n * batch,
		hipMemcpyHostToDevice
	);

	hipfftExecC2C(handle, data, data, HIPFFT_FORWARD);
	hipfftExecC2C(handle, data, data, HIPFFT_BACKWARD);

	hipDeviceProp_t devPropts;
	hipGetDeviceProperties(&devPropts, 0);
	size_t threadsPerBlock = devPropts.maxThreadsPerBlock;
	size_t blocksCount = (input.size() + threadsPerBlock - 1) / threadsPerBlock;

	normalizeKernel << <blocksCount, threadsPerBlock >> > (
		reinterpret_cast<float*>(data),
		output.size(),
		n);

	hipMemcpy(
		output.data(),
		data,
		sizeof(hipfftComplex) * n * batch,
		hipMemcpyDeviceToHost
	);

	hipfftDestroy(handle);
	hipFree(data);

	return output;
}