#include "hip/hip_runtime.h"
// Copyright (c) 2024 Kirillov Maxim
#include "fft_cufft.h"

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include ""

__global__ void normalize(float* data, int size, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] /= n;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    std::vector<float> output(input.size());

    int n = input.size() / (batch * 2);

    hipfftHandle handle;
    hipfftComplex* data;

    hipfftPlan1d(&handle, n, HIPFFT_C2C, batch);

    hipMalloc(&data, n * sizeof(hipfftComplex) * batch);
    hipMemcpy(data,input.data(), n * sizeof(hipfftComplex) * batch,hipMemcpyHostToDevice);

    hipfftExecC2C(handle, data, data, HIPFFT_FORWARD);
    hipfftExecC2C(handle, data, data, HIPFFT_BACKWARD);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
    size_t blocksCount = (input.size() + threadsPerBlock - 1) / threadsPerBlock;

    normalize<<<blocksCount, threadsPerBlock>>>(reinterpret_cast<float*>(data), output.size(), n);

    hipMemcpy(output.data(), data, n * sizeof(hipfftComplex) * batch, hipMemcpyDeviceToHost);

    hipfftDestroy(handle);
    hipFree(data);

    return output;
}
