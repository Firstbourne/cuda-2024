#include "hip/hip_runtime.h"
// Copyright (c) 2024 Tushentsova Karina
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>

__global__ void NormalizeKernel(float* input, int size, int sLen) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size){
        input[i] /= sLen;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    size_t size = input.size();
    std::vector<float> output(size);
    int sLen = size / (batch * 2);

    hipfftHandle handle;
    hipfftPlan1d(&handle, sLen, HIPFFT_C2C, batch);

    hipfftComplex* complex;
    size_t sizeBytes = sizeof(hipfftComplex) * sLen * batch;
    hipMalloc(&complex, sizeBytes);

    hipMemcpy(complex, input.data(), sizeBytes, hipMemcpyHostToDevice);

    hipfftExecC2C(handle, complex, complex, HIPFFT_FORWARD);
    hipfftExecC2C(handle, complex, complex, HIPFFT_BACKWARD);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
    size_t blocksPerGrid = (input.size() + threadsPerBlock - 1) / threadsPerBlock;

    NormalizeKernel<<<blocksPerGrid, threadsPerBlock>>>(
        reinterpret_cast<float*>(complex), size, sLen
    );

    hipMemcpy(output.data(), complex, sizeBytes, hipMemcpyDeviceToHost);

    hipfftDestroy(handle);
    hipFree(complex);

    return output;
}