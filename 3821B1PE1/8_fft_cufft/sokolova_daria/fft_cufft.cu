#include "hip/hip_runtime.h"
// Copyright (c) 2024 Sokolova Daria
#include "fft_cufft.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>

__global__ void NormalizeKernel(float* data, int totalSize, int signalSize) {
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIndex < totalSize) {
        data[threadIndex] /= signalSize;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& inputSignals, int numSignals) {
    size_t totalSize = inputSignals.size();
    std::vector<float> outputSignals(totalSize);

    int signalSize = totalSize / (numSignals * 2);

    hipfftHandle fftPlan;
    hipfftPlan1d(&fftPlan, signalSize, HIPFFT_C2C, numSignals);

    hipfftComplex* deviceData = nullptr;
    size_t bufferSize = sizeof(hipfftComplex) * signalSize * numSignals;
    hipMalloc(&deviceData, bufferSize);

    hipMemcpy(deviceData, inputSignals.data(), bufferSize, hipMemcpyHostToDevice);

    hipfftExecC2C(fftPlan, deviceData, deviceData, HIPFFT_FORWARD);
    hipfftExecC2C(fftPlan, deviceData, deviceData, HIPFFT_BACKWARD);

    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);
    size_t threadsPerBlock = deviceProperties.maxThreadsPerBlock;
    size_t blocksPerGrid = (totalSize + threadsPerBlock - 1) / threadsPerBlock;

    NormalizeKernel<<<blocksPerGrid, threadsPerBlock>>>(
        reinterpret_cast<float*>(deviceData), totalSize, signalSize
    );

    hipMemcpy(outputSignals.data(), deviceData, bufferSize, hipMemcpyDeviceToHost);

    hipfftDestroy(fftPlan);
    hipFree(deviceData);

    return outputSignals;
}
