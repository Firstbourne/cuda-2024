#include "hip/hip_runtime.h"
// Copyright (c) 2024 Morgachev Stepan
#include "fft_cufft.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>

__global__ void NormalizeKernel(float* input, int size, int signalLength) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        input[index] /= signalLength;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    size_t size = input.size();
    std::vector<float> output(size);

    int signalLength = size / (batch * 2);

    hipfftHandle handle;
    hipfftPlan1d(&handle, signalLength, HIPFFT_C2C, batch);

    hipfftComplex* complex;
    size_t sizeInBytes = sizeof(hipfftComplex) * signalLength * batch;
    hipMalloc(&complex, sizeInBytes);

    hipMemcpy(complex, input.data(), sizeInBytes, hipMemcpyHostToDevice);

    hipfftExecC2C(handle, complex, complex, HIPFFT_FORWARD);
    hipfftExecC2C(handle, complex, complex, HIPFFT_BACKWARD);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
    size_t blocksPerGrid = (input.size() + threadsPerBlock - 1) / threadsPerBlock;

    NormalizeKernel<<<blocksPerGrid, threadsPerBlock>>>(
        reinterpret_cast<float*>(complex), size, signalLength
    );

    hipMemcpy(output.data(), complex, sizeInBytes, hipMemcpyDeviceToHost);

    hipfftDestroy(handle);
    hipFree(complex);

    return output;
}
