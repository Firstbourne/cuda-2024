// Copyright (c) 2024 Kirillov Maxim
#include "gemm_cublas.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    std::vector<float> c(n * n, 0.0f);

    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_c = nullptr;

    size_t sizeInBytes = n * n * sizeof(float);

    hipMalloc(&d_a, sizeInBytes);
    hipMalloc(&d_b, sizeInBytes);
    hipMalloc(&d_c, sizeInBytes);

    hipMemcpy(d_a, a.data(), sizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), sizeInBytes, hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
            n, n, n,
            &alpha,
            d_b, HIP_R_32F, n,
            d_a, HIP_R_32F, n,
            &beta,
            d_c, HIP_R_32F, n,
            HIPBLAS_COMPUTE_32F_FAST_16F,
            HIPBLAS_GEMM_DEFAULT);

    hipMemcpy(c.data(), d_c, sizeInBytes, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}