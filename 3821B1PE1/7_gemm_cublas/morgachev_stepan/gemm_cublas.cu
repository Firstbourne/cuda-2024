// Copyright (c) 2024 Morgachev Stepan
#include "gemm_cublas.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    size_t sizeInBytes = n * n * sizeof(float);
    std::vector<float> output(n * n, 0.0f);

    float* deviceA = nullptr;
    float* deviceB = nullptr;
    float* deviceOutput = nullptr;

    hipMalloc(&deviceA, sizeInBytes);
    hipMalloc(&deviceB, sizeInBytes);
    hipMalloc(&deviceOutput, sizeInBytes);

    hipMemcpy(deviceA, a.data(), sizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, b.data(), sizeInBytes, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                deviceB, n,
                deviceA, n,
                &beta,
                deviceOutput, n);

    hipMemcpy(output.data(), deviceOutput, sizeInBytes, hipMemcpyDeviceToHost);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceOutput);
    hipblasDestroy(handle);

    return output;
}
