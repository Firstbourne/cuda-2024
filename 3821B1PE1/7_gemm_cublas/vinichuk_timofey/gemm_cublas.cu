// Copyright (c) 2024 Vinichuk Timofey
#include "gemm_cublas.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
    const std::vector<float>& b,
    int n) {

    std::vector<float> c(n * n, 0.0f);
    size_t bytesSize = n * n * sizeof(float);

    hipblasHandle_t handle;

    hipblasCreate(&handle);

    float* data_a;
    float* data_b;
    float* data_c;

    hipMalloc(&data_a, bytesSize);
    hipMalloc(&data_b, bytesSize);
    hipMalloc(&data_c, bytesSize);

    hipMemcpy(data_a, a.data(), bytesSize, hipMemcpyHostToDevice);
    hipMemcpy(data_b, b.data(), bytesSize, hipMemcpyHostToDevice);

    const float ALPHA = 1.0f;
    const float BETA = 0.0f;

    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);

    hipblasGemmEx(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, n, n,
        &ALPHA,
        data_b, HIP_R_32F, n,
        data_a, HIP_R_32F, n,
        &BETA,
        data_c, HIP_R_32F, n,
        HIPBLAS_COMPUTE_32F_FAST_16F,
        HIPBLAS_GEMM_DEFAULT
    );

    hipMemcpy(c.data(), data_c, bytesSize, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);

    hipFree(data_a);
    hipFree(data_b);
    hipFree(data_c);

    return c;
}