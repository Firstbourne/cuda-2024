// Copyright (c) 2024 Tushentsova Karina
#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a, const std::vector<float>& b, int n) {
    int totalElements = n * n;
    size_t sizeBytes = totalElements * sizeof(float);
    std::vector<float> output(totalElements, 0.0f);

    float* deviceA = nullptr;
    float* deviceB = nullptr;
    float* deviceOutput = nullptr;

    hipMalloc(&deviceA, sizeBytes);
    hipMalloc(&deviceB, sizeBytes);
    hipMalloc(&deviceOutput, sizeBytes);

    hipMemcpy(deviceA, a.data(), sizeBytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, b.data(), sizeBytes, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, deviceB, n, deviceA, n, &beta, deviceOutput, n);
    hipMemcpy(output.data(), deviceOutput, sizeBytes, hipMemcpyDeviceToHost);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceOutput);
    hipblasDestroy(handle);

    return output;
}