// Copyright (c) 2024 Sokolova Daria
#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& matrixA,
                              const std::vector<float>& matrixB,
                              int matrixDim) {
    size_t bufferSize = matrixDim * matrixDim * sizeof(float);
    std::vector<float> resultMatrix(matrixDim * matrixDim, 0.0f);

    float* deviceMatrixA = nullptr;
    float* deviceMatrixB = nullptr;
    float* deviceMatrixC = nullptr;

    hipMalloc(&deviceMatrixA, bufferSize);
    hipMalloc(&deviceMatrixB, bufferSize);
    hipMalloc(&deviceMatrixC, bufferSize);

    hipMemcpy(deviceMatrixA, matrixA.data(), bufferSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, matrixB.data(), bufferSize, hipMemcpyHostToDevice);

    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);

    const float scaleAlpha = 1.0f;
    const float scaleBeta = 0.0f;

    hipblasSgemm(cublasHandle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                matrixDim, matrixDim, matrixDim,
                &scaleAlpha,
                deviceMatrixB, matrixDim,
                deviceMatrixA, matrixDim,
                &scaleBeta,
                deviceMatrixC, matrixDim);

    hipMemcpy(resultMatrix.data(), deviceMatrixC, bufferSize, hipMemcpyDeviceToHost);

    hipFree(deviceMatrixA);
    hipFree(deviceMatrixB);
    hipFree(deviceMatrixC);
    hipblasDestroy(cublasHandle);

    return resultMatrix;
}
